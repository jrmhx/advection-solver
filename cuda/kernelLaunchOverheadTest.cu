
#include <hip/hip_runtime.h>
#include <stdio.h>
#define REPs 1e6
__global__
void test()
{
  //this is a test function.
}

int main(void)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for (int i = 0; i < REPs; ++i)
    test<<<1, 1>>>();

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Time (ms): %f\n", milliseconds/REPs);
}
